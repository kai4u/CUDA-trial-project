#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

__global__ void elementwiseAnd(int* a, int* b, int* res){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	res[index] = a[index] && b[index];
}

void initMatrix(int* a, size_t n_row, size_t n_col){	

    std::default_random_engine gen;
	std::bernoulli_distribution dist(0.5);    


	for(size_t i = 0; i < n_row; i++)
		for(size_t j = 0; j < n_col; j++){
			size_t index = j + i * n_col;
			a[index] = (dist(gen));
		}
}

int main(){
	
	using namespace std;
	
	size_t n_row, n_col;
	cin >> n_row >> n_col;
    size_t size = n_row * n_col * sizeof(int);
	
	int *c, *res;
    a = (bool *) malloc(size);initMatrix(a, n_row, n_col);
    b = (bool *) malloc(size);initMatrix(b, n_row, n_col);
    c = (int *) malloc(size);
    res = (int *) malloc(size);
    
	for(size_t i = 0; i < n_row; i++)
		for(size_t j = 0; j < n_col; j++){
			size_t index = j + i * n_col;
			c[index] = a[index] && b[index];
		}
    
   
    cout << endl;
    
	int *d_a, *d_b, *d_c;	

	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	HANDLE_ERROR(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));

	elementwiseAnd<<<n_col, n_row>>>(d_a, d_b, d_c);

	hipMemcpy(res, d_c, size, hipMemcpyDeviceToHost);

	int loss = 0;

    for(size_t i = 0; i < n_row; i++){
		for(size_t j = 0; j < n_col; j++){
			size_t index = j + i * n_col;
			loss += abs(c[index] - res[index]);
		}
    }
	cout << loss << endl;
    
    free(a);
    free(b);
    free(c);
    free(res);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	return 0;
}
